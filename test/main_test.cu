#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <algorithm>
#include <type_traits>

constexpr int blocksize = 16;

__global__ void fillStuff(int *in) {
	in[threadIdx.x] = threadIdx.x;
}

__global__ void runStuff(int *in, int *out) {
	out[threadIdx.x] = in[threadIdx.x] * 2;
}

TEST(MainCudaTests, CudaWorks) {
	dim3 block{blocksize, 1};
	dim3 grid{1, 1};


	int *in, *out;

	hipMalloc((void**)&in, blocksize * sizeof(int));
	hipMalloc((void**)&out, blocksize * sizeof(int));

	fillStuff<<<grid, block>>>(in);
	runStuff<<<grid, block>>>(in, out);

	int result[blocksize];

	hipMemcpy(result, out, blocksize, hipMemcpyHostToDevice);

	hipFree(in);
	hipFree(out);
}